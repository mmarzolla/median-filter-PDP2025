#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-median-filter-2D.cu -- GPU median filter using multilevel histograms
 *
 * Copyright (C) 2023, 2025 Moreno Marzolla, Michele Ravaioli
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ****************************************************************************/

#include "hpc.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <stdint.h>

#include "common.h"

// threads per CUDA block
#define BLKDIM 1024
// threads per CUDA 2D block (BLKDIM_2D * BLKDIM_2D)
#define BLKDIM_2D 32
// CUDA warp size
#define WARP_SIZE 32
// number of warps per block
#define NUM_WARPS (BLKDIM / WARP_SIZE)

#define HIST_SIZE 256

/* The following kernel assumes that threads are organized into a 2D
   grid, and that there are as many threads as pixels in the extended
   image, i.e., image with ghost area.

   Image `in` must have size width * height;

   Image `out` must have size (width + 2*radius) * (height + 2*radius)

   `width` and `height` represent the size of the input image, and
   must not include the ghost area.
*/
static __global__
void init_ghost_area( const data_t *in, data_t *out, int width, int height, int radius )
{
    // These coordinates refer to the output image (with ghost area)
    const int DST_X = threadIdx.x + blockIdx.x * blockDim.x;
    const int DST_Y = threadIdx.y + blockIdx.y * blockDim.y;
    const int ext_width = width + 2*radius;
    const int ext_height = height + 2*radius;

    if (DST_X >= ext_width || DST_Y >= ext_height)
        return;

    // These coordinates refer to the input image (without ghost area)
    int SRC_X = DST_X - radius, SRC_Y = DST_Y - radius;

    if (SRC_X < 0)
        SRC_X = 0;
    else if (SRC_X >= width)
        SRC_X = width-1;

    if (SRC_Y < 0)
        SRC_Y = 0;
    else if (SRC_Y >= height)
        SRC_Y = height-1;

    out[DST_X + DST_Y*ext_width] = in[SRC_X + SRC_Y*width];
}

/**
 * All threads in the same warp cooperate to zero histogram `hist[]`
 * of length `HIST_SIZE`.
 */
static __device__
void warp_init_hist( int* hist, int lane_id )
{
#pragma unroll
    for (int i=lane_id; i<HIST_SIZE; i+=WARP_SIZE) {
        hist[i] = 0;
    }
}

static __global__
void median_filter_kernel_generic( const data_t* in,
                                   data_t* out,
                                   const int WIDTH,
                                   const int HEIGHT,
                                   const int radius )
{
    // ID of the current warp
    const int WARP_ID = threadIdx.x / WARP_SIZE;
    // ID of the thread within the warp (0 <= LAND_ID < WARP_SIZE)
    const int LANE_ID = threadIdx.x % WARP_SIZE;

    // Histograms in shared memory
    __shared__ int sh_hist[NUM_WARPS][HIST_SIZE];

    // the histogram handled by the current warp
    int *warp_hist = &sh_hist[WARP_ID][0];

    const int NPASSES = DATA_SIZE;

    __shared__ data_t mask[NUM_WARPS];
    __shared__ data_t key[NUM_WARPS];
    __shared__ int median_pos[NUM_WARPS];
    __shared__ int shift_amount[NUM_WARPS];

    // Pixel coordinates
    const int pixelX = threadIdx.x / WARP_SIZE + blockIdx.x * NUM_WARPS;
    const int pixelY = threadIdx.y + blockIdx.y * blockDim.y;

    if (pixelY >= HEIGHT || pixelX >= WIDTH)
        return;

    const int WINDOW_L = (2 * radius) + 1;
    const int WINDOW_SIZE = WINDOW_L * WINDOW_L;
    const int EXT_WIDTH = (2 * radius) + WIDTH;

    if (0 == LANE_ID) {
        shift_amount[WARP_ID] = 8*(NPASSES - 1);
        mask[WARP_ID] = 0;
        key[WARP_ID] = 0;
        median_pos[WARP_ID] = WINDOW_SIZE / 2;
    }
    __syncwarp();

#pragma unroll
    for (int pass=0; pass<NPASSES; pass++) {
        // Init histogram
        warp_init_hist(warp_hist, LANE_ID);
        __syncwarp();

        // Fill histogram; all threads in this warp contribute to
        // filling the histogram of the filter region centered at
        // (pixelX, pixelY)
        for (int i=LANE_ID; i<WINDOW_SIZE; i+=WARP_SIZE) {
            // window pixel coords
            const int win_pX = (i % WINDOW_L) + pixelX;
            const int win_pY = (i / WINDOW_L) + pixelY;

            const data_t val = in[win_pX + (win_pY * EXT_WIDTH)];
            if ((val & mask[WARP_ID]) == key[WARP_ID]) {
                const int idx = (val >> shift_amount[WARP_ID]) & 0xff;
                atomicAdd(&warp_hist[idx], 1);
            }
        }
        __syncwarp();

        // Search median from histogram (only the master of each warp)
        if (LANE_ID == 0) {
            int k=0;
            for (k=0; (k<HIST_SIZE-1) && (median_pos[WARP_ID] >= warp_hist[k]); k++)
                median_pos[WARP_ID] -= warp_hist[k];
            key[WARP_ID] |= k << shift_amount[WARP_ID];
            mask[WARP_ID] |= 0xff << shift_amount[WARP_ID];
            shift_amount[WARP_ID] -= 8;
        }
        __syncwarp();
    }

    if (0 == LANE_ID) {
        out[pixelX + (pixelY * WIDTH)] = key[WARP_ID];
    }
}

extern "C"
void cuda_median_2D_hist_generic( const data_t *in, data_t *out,
                                  const int *dims, int ndims, int radius )
{
    data_t *d_in, *d_out;

    assert(ndims == 2);
    const int width = dims[DX];
    const int height = dims[DY];

    const int EXT_WIDTH = (2 * radius) + width;
    const int EXT_HEIGHT = (2 * radius) + height;

    const size_t SIZE = width * height * DATA_SIZE;
    const size_t EXT_SIZE = EXT_WIDTH * EXT_HEIGHT * DATA_SIZE;

    cudaSafeCall( hipMalloc((void**)&d_in, EXT_SIZE) );
    cudaSafeCall( hipMalloc((void**)&d_out, SIZE) );

    // Initialize the ghost area
    cudaSafeCall( hipMemcpy(d_out, in, SIZE, hipMemcpyHostToDevice) );
    const dim3 INIT_BLOCK(BLKDIM_2D, BLKDIM_2D);
    const dim3 INIT_GRID((EXT_WIDTH + BLKDIM_2D - 1) / BLKDIM_2D,
                         (EXT_HEIGHT + BLKDIM_2D - 1) / BLKDIM_2D);
    init_ghost_area<<< INIT_GRID, INIT_BLOCK >>>(d_out, d_in, width, height, radius);
    cudaCheckError();

    // Start computation
    const dim3 GRID((width + NUM_WARPS - 1) / NUM_WARPS, height);
    median_filter_kernel_generic<<< GRID, BLKDIM >>>(d_in, d_out, width, height, radius);
    cudaCheckError();
    cudaSafeCall( hipMemcpy(out, d_out, SIZE, hipMemcpyDeviceToHost) );

    hipFree(d_in);
    hipFree(d_out);
}
